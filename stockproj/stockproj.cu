#include "hip/hip_runtime.h"
#include "stockrun.cuh"
#include "Shlwapi.h"

#define nIter 5

size_t initialTrainSamples = 2;
size_t trainSamplesIncreaseFactor = 2;
float trainIncreaseThreshold = 100.0f;

float initialStepMult = 40.0f;
float minimumStepMult = 0.5f;
float stepMultDecFactor = 0.707f;
float annealingStartError = 500.0f;

size_t backupInterval = 0;
size_t backupSampleNumStart = 0;

#define datastring "rawdata/"
#define savestring "saveweights/"

#define RAND_EXPLICIT			//uses a test set with inputs in a random order (specified by randtrainstring)

void saveResults(size_t numRuns, float afterError);
void saveSetHistory(size_t nSamples, size_t nRuns, float stepFacMult);
void loadLocalParameters();
void loadSimVariables();
void saveSimVariables();
size_t readData(size_t begin, size_t numIOs);
void backupFiles(std::string backname);

size_t numRuns;
size_t trainSamples;
size_t totalSamples;
size_t numRunSetStart;

float stepMult;
float stepAdjustment = 1.0f;
size_t stepAdjustmentNumStart = 0.0f;

float redoErrorThreshold = 0.0f;
float redoStepAdjustment = 1.0f;
float successStepAdjustment = 1.0f;

bool randomizeTrainSetEveryRun = true;
bool randomizeSubsetOnThreshold = false;

bool pairedTraining = false;

#define ERRORS_SAVED 5
std::list<float> lastErrors;

float annealingMultiplier() {
	if (lastErrors.size() == 0 || annealingStartError == 0)
		return 1;

	float avg = 0;
	for (std::list<float>::const_iterator it = lastErrors.begin(); it != lastErrors.end(); it++) {
		avg += *it;
	}
	avg /= lastErrors.size();

	if (avg > annealingStartError)
		return 1;
	return avg / annealingStartError;
}

float stepMultiplier(size_t numRuns) {
	return annealingMultiplier()*stepMult*stepAdjustment;
}

void updateLastErrors(float error) {
	lastErrors.push_back(error);
	if (lastErrors.size() > ERRORS_SAVED)
		lastErrors.pop_front();
}


int main() {
	srand((size_t)time(NULL));

	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, 0));
	if (!prop.canMapHostMemory)
		exit(0);
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

	loadParameters("pars.cfg");
	loadLocalParameters();

	setStrings(datastring, savestring);
	numRuns = 0;
	trainSamples = initialTrainSamples;
	stepMult = initialStepMult;
	numRunSetStart = 0;

	loadSimVariables();

	LayerCollection layers;
	PairedConvCollection pairedLayers;
	if (pairedTraining) {
		pairedLayers = createAndInitializePairedConvCollection(NUM_INPUTS);
		loadPairedWeights(pairedLayers, savename);
	}
	else {
		layers = createLayerCollection(0, getLCType());
		initializeLayers(&layers);
		loadWeights(layers, savename);
	}

	size_t numSamples;
	if (randomizeSubsetOnThreshold) {
		size_t tSamples = readData(1, 0);
		randomizeTrainSet();
		numSamples = min(trainSamples, tSamples);
	}
	else {
		numSamples = readData(1, trainSamples);
	}

	std::cout << "Calculating initial error: ";
	float initError;
	float* initSecError = new float[5];
	initSecError[0] = 0.0f;
	initSecError[1] = 0.0f;
	initSecError[2] = 0.0f;
	initSecError[3] = 0.0f;
	initSecError[4] = 0.0f;
	auto initstart = std::chrono::high_resolution_clock::now();
	if (pairedTraining)
		initError = runPairedSim(pairedLayers, false, 0, trainSamples);
	else {
		initError = runSim(layers, false, 0, trainSamples, false, initSecError);
	}
	float prevAfterError = initError;

	auto initelapsed = std::chrono::high_resolution_clock::now() - initstart;
	long long inittime = std::chrono::duration_cast<std::chrono::microseconds>(initelapsed).count();

	std::cout << inittime / 1000000 << " s, Error: " << initError;
	if (initSecError[0] != 0.0f)
		std::cout << " SE1: " << initSecError[0];
	if (initSecError[1] != 0.0f)
		std::cout << " SE2: " << initSecError[1];
	if (initSecError[2] != 0.0f)
		std::cout << " SE3: " << initSecError[2];
	if (initSecError[3] != 0.0f)
		std::cout << " SE4: " << initSecError[3];
	if (initSecError[4] != 0.0f)
		std::cout << " SE5: " << initSecError[4];
	std::cout << std::endl;
	delete[] initSecError;
	updateLastErrors(initError);

	while (true) {
		if (pairedTraining)
			std::cout << nIter << "+1 runs on " << numSamples << " sample pairs";
		else
			std::cout << nIter << "+1 runs on " << numSamples << " samples";
		if (trainSamples >= stepAdjustmentNumStart && redoErrorThreshold > 0.0f) {
			std::cout << "(SA: " << stepAdjustment << ")";
		}
		std::cout << ": ";
		auto gpustart = std::chrono::high_resolution_clock::now();

		if (randomizeSubsetOnThreshold) {
			randomizeTrainSet(trainSamples);
		}
		else if (randomizeTrainSetEveryRun)
			randomizeTrainSet();

		for (size_t i = 0; i < nIter; i++) {
			if (pairedTraining)
				runPairedSim(pairedLayers, true, stepMultiplier(numRuns), trainSamples);
			else
				runSim(layers, true, stepMultiplier(numRuns), trainSamples);
#ifdef BATCH_MODE
			if (pairedTraining) {
				batchUpdate(pairedLayers.conv1);
				batchUpdate(pairedLayers.conv2);
				batchUpdate(pairedLayers.fixed);
			}
			else
				batchUpdate(layers);
#endif
		}

		float afterError;
		float* afterSecError = new float[5];
		afterSecError[0] = 0.0f;
		afterSecError[1] = 0.0f;
		afterSecError[2] = 0.0f;
		afterSecError[3] = 0.0f;
		afterSecError[4] = 0.0f;
		if (pairedTraining)
			afterError = runPairedSim(pairedLayers, false, 0, trainSamples);
		else
			afterError = runSim(layers, false, 0, trainSamples, false, afterSecError);

		auto gpuelapsed = std::chrono::high_resolution_clock::now() - gpustart;
		long long gputime = std::chrono::duration_cast<std::chrono::microseconds>(gpuelapsed).count();
		std::cout << gputime / 1000000 << " s, Error: " << afterError;
		if (afterSecError[0] != 0.0f) {
			std::cout << " SE1: " << afterSecError[0];
		}
		if (afterSecError[1] != 0.0f) {
			std::cout << " SE2: " << afterSecError[1];
		}
		if (afterSecError[2] != 0.0f) {
			std::cout << " SE3: " << afterSecError[2];
		}
		if (afterSecError[3] != 0.0f) {
			std::cout << " SE4: " << afterSecError[3];
		}
		if (afterSecError[4] != 0.0f) {
			std::cout << " SE5: " << afterSecError[4];
		}
		std::cout << std::endl;
		delete[] afterSecError;

		if (trainSamples >= stepAdjustmentNumStart) {
			if (redoErrorThreshold > 0.0f && prevAfterError > 0.0f && afterError - prevAfterError > redoErrorThreshold) {
				std::cout << "Error increase was above threshold; redoing last run with lower stepfactor" << std::endl;

				if (pairedTraining)
					loadPairedWeights(pairedLayers, savename);
				else
					loadWeights(layers, savename);
				stepAdjustment *= redoStepAdjustment;
				continue;
			}
			stepAdjustment *= successStepAdjustment;
		}
		prevAfterError = afterError;
		updateLastErrors(afterError);
		if (pairedTraining)
			savePairedWeights(pairedLayers, savename);
		else
			saveWeights(layers, savename);
		numRuns += nIter;

		if (afterError < trainIncreaseThreshold && trainSamples < totalSamples) {
			saveSetHistory(numSamples, numRuns - numRunSetStart, stepMult);
			numRunSetStart = numRuns;
			trainSamples = min(trainSamplesIncreaseFactor * trainSamples, totalSamples);
			if (randomizeSubsetOnThreshold) {
				randomizeTrainSet();
				numSamples = trainSamples;
				std::cout << "Starting new run on " << numSamples << " samples" << std::endl;
			}
			else
				numSamples = readData(1, trainSamples);
			stepMult = max(stepMultDecFactor*stepMult, minimumStepMult);
			prevAfterError = -1.0f;
		}

		saveResults(numRuns, afterError);
		saveSimVariables();

		if (trainSamples >= backupSampleNumStart && backupInterval > 0 && (numRuns - numRunSetStart) % backupInterval == 0) {
			std::stringstream bss;
			bss << savename << numSamples << "-" << numRuns - numRunSetStart;
			backupFiles(bss.str().c_str());
		}
	}
}

void saveResults(size_t numRuns, float afterError) {
	std::stringstream resname;
	resname << savestring << savename << "result";
	std::ofstream resfile(resname.str().c_str(), std::ios_base::app);
	resfile << numRuns << " " << afterError << std::endl;
}

void saveSetHistory(size_t nSamples, size_t nRuns, float stepFacMult) {
	std::stringstream hisname;
	hisname << savestring << savename << "history";
	std::ofstream hisfile(hisname.str().c_str(), std::ios_base::app);
	hisfile << nSamples << " " << stepFacMult << " " << nRuns << std::endl;
}

void loadSimVariables() {
	std::stringstream pss;
	pss << savestring << savename << "pars";
	std::ifstream infile(pss.str().c_str());

	std::string line;
	while (getline(infile, line)) {
		std::stringstream lss(line);
		std::string var;
		lss >> var;
		if (var == "numRuns")
			lss >> numRuns;
		if (var == "trainSamples")
			lss >> trainSamples;
		if (var == "stepMult")
			lss >> stepMult;
		if (var == "numRunSetStart")
			lss >> numRunSetStart;
	}
}

void saveSimVariables() {
	std::stringstream pss;
	pss << savestring << savename << "pars";
	std::ofstream outfile(pss.str().c_str());

	outfile << "numRuns " << numRuns << std::endl;
	outfile << "trainSamples " << trainSamples << std::endl;
	outfile << "stepMult " << stepMult << std::endl;
	outfile << "numRunSetStart " << numRunSetStart << std::endl;
}

size_t readData(size_t begin, size_t numIOs) {
	if (numIOs > 0)
		std::cout << "Reading " << numIOs << " samples from trainset: ";
	else
		std::cout << "Reading all samples from trainset: ";

	auto readstart = std::chrono::high_resolution_clock::now();
#ifndef RAND_EXPLICIT
	totalSamples = readTrainSet(trainstring, begin, numIOs);
#else
	totalSamples = readExplicitTrainSet(randtrainstring, begin, numIOs);
#endif
	auto readelapsed = std::chrono::high_resolution_clock::now() - readstart;
	long long readtime = std::chrono::duration_cast<std::chrono::microseconds>(readelapsed).count();
	std::cout << readtime / 1000000 << " s" << std::endl;
	size_t numSamples;
	if (numIOs > 0)
		numSamples = min(numIOs, totalSamples);
	else
		numSamples = totalSamples;
	std::cout << numSamples << "/" << totalSamples << " samples loaded" << std::endl;
	return numSamples;
}

void loadLocalParameters() {
	std::ifstream infile("pars.cfg");
	std::string line;
	while (getline(infile, line)) {
		std::stringstream lss(line);
		std::string var;
		lss >> var;

		if (var == "initialStepMult")
			lss >> initialStepMult;
		else if (var == "minimumStepMult")
			lss >> minimumStepMult;
		else if (var == "stepMultDecFactor")
			lss >> stepMultDecFactor;
		else if (var == "annealingStartError")
			lss >> annealingStartError;
		else if (var == "initialTrainSamples")
			lss >> initialTrainSamples;
		else if (var == "trainSamplesIncreaseFactor")
			lss >> trainSamplesIncreaseFactor;
		else if (var == "trainIncreaseThreshold")
			lss >> trainIncreaseThreshold;
		else if (var == "backupInterval")
			lss >> backupInterval;
		else if (var == "backupSampleNumStart")
			lss >> backupSampleNumStart;
		else if (var == "randomizeSubsetOnThreshold")
			lss >> randomizeSubsetOnThreshold;
		else if (var == "randomizeTrainSetEveryRun")
			lss >> randomizeTrainSetEveryRun;
		else if (var == "pairedTraining")
			lss >> pairedTraining;
		else if (var == "redoErrorThreshold")
			lss >> redoErrorThreshold;
		else if (var == "redoStepAdjustment")
			lss >> redoStepAdjustment;
		else if (var == "successStepAdjustment")
			lss >> successStepAdjustment;
		else if (var == "stepAdjustmentNumStart")
			lss >> stepAdjustmentNumStart;
	}
}

void backupFiles(std::string backname) {
	std::stringstream bss;
	bss << savestring << "backup/" << backname;

	std::stringstream oss;
	oss << savestring << savename;

	std::stringstream pss;
	pss << oss.str();

	std::stringstream nss;
	nss << bss.str();

	CopyFile(pss.str().c_str(), nss.str().c_str(), false);

	if(pairedTraining) {
		pss.clear();
		pss.str("");
		nss.clear();
		nss.str("");

		pss << oss.str() << "conv";
		nss << bss.str() << "conv";

		CopyFile(pss.str().c_str(), nss.str().c_str(), false);

		pss.clear();
		pss.str("");
		nss.clear();
		nss.str("");

		pss << oss.str() << "fixed";
		nss << bss.str() << "fixed";

		CopyFile(pss.str().c_str(), nss.str().c_str(), false);
	}

	pss.clear();
	pss.str("");
	nss.clear();
	nss.str("");

	pss << oss.str() << "result";
	nss << bss.str() << "result";

	CopyFile(pss.str().c_str(), nss.str().c_str(), false);

	pss.clear();
	pss.str("");
	nss.clear();
	nss.str("");

	pss << oss.str() << "pars";
	nss << bss.str() << "pars";

	CopyFile(pss.str().c_str(), nss.str().c_str(), false);

	pss.clear();
	pss.str("");
	nss.clear();
	nss.str("");

	pss << oss.str() << "history";
	nss << bss.str() << "history";

	CopyFile(pss.str().c_str(), nss.str().c_str(), false);
}