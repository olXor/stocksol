#include "hip/hip_runtime.h"
#include <stockrun.cuh>
#include "Shlwapi.h"

#ifdef LOCAL
#define datastring "rawdata/"
#define savestring "saveweights/"
#else
#define datastring "../stockproj/rawdata/"
#define savestring "../stockproj/saveweights/"
#endif

bool testExplicitFile = false;
size_t testBegin = 1;
size_t testNumIOs = 0;

bool testUseSampleFile = false;
bool discardSamples = false;
std::string testOutputFile = "testoutput";
std::string testfile = "trainset";

bool testPrintSampleAll = false;

size_t readData(size_t begin, size_t numIOs);
void loadLocalParameters(std::string parName);

int main() {
	srand((size_t)time(NULL));
#ifdef LOCAL
	loadLocalParameters("pars.cfg");
	loadParameters("pars.cfg");
#else
	loadLocalParameters("../stockproj/pars.cfg");
	loadParameters("../stockproj/pars.cfg");
#endif
	setStrings(datastring, savestring);

	LayerCollection layers = createLayerCollection();
	initializeLayers(&layers);
	loadWeights(layers, savename);

	readData(testBegin, testNumIOs);

	if (testUseSampleFile)
		sampleTestSim(layers, testOutputFile, testPrintSampleAll);
	else
		runSim(layers, false, 0, 0, true);

#ifdef LOCAL
	system("pause");
#endif
}

size_t readData(size_t begin, size_t numIOs) {
	size_t numSamples;
	if (!testUseSampleFile) {
		if (numIOs > 0)
			std::cout << "Reading " << numIOs << " samples from trainset: ";
		else
			std::cout << "Reading all samples from trainset: ";

		auto readstart = std::chrono::high_resolution_clock::now();
		size_t totalSamples;
		if (!testExplicitFile)
			totalSamples = readTrainSet(testfile, begin, numIOs);
		else
			totalSamples = readExplicitTrainSet(testfile, begin, numIOs);

		auto readelapsed = std::chrono::high_resolution_clock::now() - readstart;
		long long readtime = std::chrono::duration_cast<std::chrono::microseconds>(readelapsed).count();
		std::cout << readtime / 1000000 << " s" << std::endl;
		size_t numSamples;
		if (numIOs > 0)
			numSamples = min(numIOs, totalSamples);
		else
			numSamples = totalSamples;
		std::cout << numSamples << "/" << totalSamples << " samples loaded" << std::endl;
		return numSamples;
	}
	else {
		std::cout << "Reading trainset: ";
		auto readstart = std::chrono::high_resolution_clock::now();
		size_t numDiscards[2];

		sampleReadTrainSet(testfile, discardSamples, numDiscards);

		auto readelapsed = std::chrono::high_resolution_clock::now() - readstart;
		long long readtime = std::chrono::duration_cast<std::chrono::microseconds>(readelapsed).count();
		std::cout << readtime / 1000000 << " s" << std::endl;

		std::cout << numDiscards[0] << "/" << numDiscards[1] << " samples discarded" << std::endl;

		numSamples = numDiscards[1] - numDiscards[0];
	}
	return numSamples;
}

void loadLocalParameters(std::string parName) {
	std::ifstream infile(parName.c_str());
	std::string line;
	while (getline(infile, line)) {
		std::stringstream lss(line);
		std::string var;
		lss >> var;
		if (var == "testUseSampleFile")
			lss >> testUseSampleFile;
		else if (var == "testBegin")
			lss >> testBegin;
		else if (var == "testNumIOs")
			lss >> testNumIOs;
		else if (var == "discardSamples")
			lss >> discardSamples;
		else if (var == "testOutputFile")
			lss >> testOutputFile;
		else if (var == "testExplicitFile")
			lss >> testExplicitFile;
		else if (var == "testfile")
			lss >> testfile;
		else if (var == "testPrintSampleAll")
			lss >> testPrintSampleAll;
	}
}
