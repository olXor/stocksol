#include "hip/hip_runtime.h"
#include "stockrun.cuh"
#include <random>

#ifdef LOCAL
#define datastring "rawdata/"
#define savestring "saveweights/"
#else
#define datastring "../stockproj/rawdata/"
#define savestring "../stockproj/saveweights/"
#endif

std::string selecttestfile = "testfile";

struct SelectionCriteria {
	size_t minSubnetSelect;
	size_t oppositeMinSubnetSelect;
	std::vector<float> testSelectBinMins;
	std::vector<float> testSelectBinMaxes;
	std::vector<float> oppositeSelectBinMins;
	std::vector<float> oppositeSelectBinMaxes;
};

std::vector < std::vector<std::vector<float>>> subnetResults;	//[dataset.size(), 2*numSubnets, numBins]

bool testExplicitFile = false;
size_t testBegin = 1;
size_t testNumIOs = 0;

bool testUseSampleFile = false;
bool discardSamples = false;

void loadLocalParameters(std::string parName);

size_t numSubnets = 1;
size_t minSubnetSelect = 1;	//the number of subnets that need to return a positive result to select
size_t oppositeMinSubnetSelect = 1;

std::vector<LayerCollection> longsubnets;
std::vector<LayerCollection> shortsubnets;

bool longSelection = true;
bool shortSelection = false;
bool randomizeSelectOptimizationStartPoint = true;

bool selectBasedOnMaxBin = false;
size_t binToSelectOn = 0;

size_t selectionEvaluationType = 0;
float selectionEvaluationProfitPower = 2.0f;

float selectPerturbSigma = 2.0f;

size_t readData(std::string fname, size_t begin, size_t numIOs);
float evaluateSelectionCriteria(SelectionCriteria crit, bool print);
SelectionCriteria getRandomSelectionCriteria();
SelectionCriteria perturbSelectionCriteria(SelectionCriteria crit);

void saveSelectionCriteria(SelectionCriteria crit);
bool loadSelectionCriteria(SelectionCriteria* crit);
void printSelectionCriteria(SelectionCriteria crit);
void generateSubnetResults();
void evaluateMaxBinSelection(size_t bin, size_t numSubnetsToSelect, bool print);

size_t numSelectCrossValSets = 0;

bool selectOnMinBackupWeights = true;

int main() {
	srand((size_t)time(NULL));

	hipDeviceProp_t prop;
	checkCudaErrors(hipGetDeviceProperties(&prop, 0));
	if (!prop.canMapHostMemory)
		exit(0);
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

#ifdef LOCAL
	loadLocalParameters("pars.cfg");
	loadParameters("pars.cfg");
#else
	loadLocalParameters("../stockproj/pars.cfg");
	loadParameters("../stockproj/pars.cfg");
#endif

	setStrings(datastring, savestring);


	longsubnets.resize(numSubnets);
	shortsubnets.resize(numSubnets);
	for (size_t cv = 0; cv < numSelectCrossValSets || numSelectCrossValSets == 0; cv++) {
		std::string fname = selecttestfile;
		if (numSelectCrossValSets != 0) {
			std::stringstream fss;
			fss << selecttestfile << cv + 1;
			fname = fss.str();
		}
		readData(fname, testBegin, testNumIOs);
		std::cout << "Loading subnets for CV set " << cv + 1 << ": ";
		for (size_t i = 0; i < numSubnets; i++) {
			if (longSelection) {
				longsubnets[i] = createLayerCollection(0, getLCType());
				initializeLayers(&longsubnets[i]);

				std::stringstream wss;
				if (selectOnMinBackupWeights)
					wss << "backup/";
				wss << savename;
				wss << "long";
				if (numSelectCrossValSets != 0)
					wss << cv + 1 << "-";
				wss << i + 1;
				if (selectOnMinBackupWeights)
					wss << "Min";
				//std::cout << "Loading subnet " << wss.str().c_str() << std::endl;
				if (!loadWeights(longsubnets[i], wss.str().c_str())) {
					std::cout << "couldn't find long weights file #" << i + 1 << std::endl;
#ifdef LOCAL
					system("pause");
#endif
					return 0;
				}
			}
			if (shortSelection) {
				shortsubnets[i] = createLayerCollection(0, getLCType());
				initializeLayers(&shortsubnets[i]);

				std::stringstream wss;
				if (selectOnMinBackupWeights)
					wss << "backup/";
				wss << savename;
				wss << "short";
				if (numSelectCrossValSets != 0)
					wss << cv + 1 << "-";
				wss << i + 1;
				if (selectOnMinBackupWeights)
					wss << "Min";
				//std::cout << "Loading subnet " << wss.str().c_str() << std::endl;
				if (!loadWeights(shortsubnets[i], wss.str().c_str())) {
					std::cout << "couldn't find short weights file #" << i + 1 << std::endl;
#ifdef LOCAL
					system("pause");
#endif
					return 0;
				}
			}
		}
		std::cout << "done" << std::endl;

		SelectionCriteria currentCrit;
		if (!loadSelectionCriteria(&currentCrit)) {
			if (randomizeSelectOptimizationStartPoint)
				currentCrit = getRandomSelectionCriteria();
			else {
				currentCrit.minSubnetSelect = minSubnetSelect;
				currentCrit.oppositeMinSubnetSelect = oppositeMinSubnetSelect;
				currentCrit.testSelectBinMins = testSelectBinMins;
				currentCrit.testSelectBinMaxes = testSelectBinMaxes;
				currentCrit.oppositeSelectBinMins = oppositeSelectBinMins;
				currentCrit.oppositeSelectBinMaxes = oppositeSelectBinMaxes;
			}
		}

		std::cout << "Generating subnet results";
		if (numSelectCrossValSets != 0)
			std::cout << " for CV set " << cv+1;
		std::cout << ": ";
		auto genstart = std::chrono::high_resolution_clock::now();
		generateSubnetResults();
		auto genelapsed = std::chrono::high_resolution_clock::now() - genstart;
		long long gentime = std::chrono::duration_cast<std::chrono::microseconds>(genelapsed).count();
		std::cout << " (" << gentime / 1000000 << " s)" << std::endl;

		if (selectBasedOnMaxBin) {
			std::cout << "Printing results by required number of agreeing bins." << std::endl;
			for (size_t i = 0; i <= numSubnets; i++) {
				evaluateMaxBinSelection(binToSelectOn, i, true);
			}
		}
		else {
			float currentBest = -999999.0f;
			SelectionCriteria testCrit = currentCrit;
			while (true) {
				float testEval = evaluateSelectionCriteria(testCrit, false);

				if (testEval > currentBest || currentBest == -999999.0f) {
					currentBest = testEval;
					currentCrit = testCrit;
					printSelectionCriteria(currentCrit);
					evaluateSelectionCriteria(currentCrit, true);
					saveSelectionCriteria(currentCrit);
				}
				testCrit = perturbSelectionCriteria(currentCrit);
			}
		}
	}

#ifdef LOCAL
	system("pause");
#endif
}

void generateSubnetResults() {
	std::vector<float*> d_inputs(2*numSubnets);
	std::vector<float*> h_output(2*numSubnets), d_output(2*numSubnets);
	for (size_t i = 0; i < 2*numSubnets; i++) {
		LayerCollection layers;
		size_t subnetPos = i % numSubnets;
		if (i < numSubnets)
			layers = longsubnets[subnetPos];
		else
			layers = shortsubnets[subnetPos];

		if (layers.numConvolutions > 0) {
			if (layers.convPars[0].numInputLocs != NUM_INPUTS || layers.convPars[0].numInputNeurons != 1)
				throw std::runtime_error("inputs to first layer don't match data set");
			d_inputs[i] = layers.convMat[0].inlayer;
		}
		else if (layers.numFixedNets > 0) {
			if (layers.fixedPars[0].numInputNeurons != NUM_INPUTS)
				throw std::runtime_error("inputs to first layer don't match data set");
			d_inputs[i] = layers.fixedMat[0].inlayer;
		}
		else
			throw std::runtime_error("tried to run on a network with no convolutions and no fixed networks");

		checkCudaErrors(hipHostAlloc(&h_output[i], numBins*sizeof(float), hipHostMallocMapped));
		checkCudaErrors(hipHostGetDevicePointer(&d_output[i], h_output[i], 0));

		disableDropout();
		generateDropoutMask(&layers);
	}

	hipStream_t mainStream = 0;

	std::vector<IOPair>* dataset = getTrainSet();
	subnetResults.resize(dataset->size());
	for (size_t i = 0; i < subnetResults.size(); i++) {
		subnetResults[i].resize(2 * numSubnets);
		for (size_t j = 0; j < subnetResults[i].size(); j++) {
			subnetResults[i][j].resize(numBins);
		}
	}

	for (size_t i = 0; i < dataset->size(); i++) {
		for (size_t j = 0; j < 2 * numSubnets; j++) {
			LayerCollection layers;
			size_t subnetPos = j % numSubnets;
			if (j < numSubnets)
				layers = longsubnets[subnetPos];
			else
				layers = shortsubnets[subnetPos];

			checkCudaErrors(hipMemcpyAsync(d_inputs[j], &(*dataset)[i].inputs[0], NUM_INPUTS*sizeof(float), hipMemcpyHostToDevice, mainStream));

			calculate(layers, mainStream);

			checkCudaErrors(hipMemcpyAsync(layers.correctoutput, &(*dataset)[i].correctbins[0], numBins*sizeof(float), hipMemcpyHostToDevice, mainStream));

			calculateOutputError << <1, numBins, 0, mainStream >> >(layers.d_fixedMat[layers.numFixedNets - 1], layers.stepfactor, layers.correctoutput, d_output[j]);
		}
		checkCudaErrors(hipDeviceSynchronize());
		for (size_t j = 0; j < 2 * numSubnets; j++)
			for (size_t k = 0; k < numBins; k++)
				subnetResults[i][j][k] = h_output[j][k];
	}
}

float evaluateSelectionCriteria(SelectionCriteria crit, bool print) {
	std::vector<IOPair>* dataset = getTrainSet();
	float longProfit = 0.0f;
	float shortProfit = 0.0f;
	size_t numLongTrades = 0;
	size_t numShortTrades = 0;
	std::vector<size_t> longDist(numBins);
	std::vector<size_t> shortDist(numBins);
	for (size_t i = 0; i < subnetResults.size(); i++) {
		size_t numLongTestSelected = 0;
		size_t numLongOppositeSelected = 0;
		size_t numShortTestSelected = 0;
		size_t numShortOppositeSelected = 0;
		for (size_t j = 0; j < 2 * numSubnets; j++) {
			bool testSelected = true;
			bool oppositeSelected = true;
			for (size_t k = 0; k < numBins; k++) {
				if (subnetResults[i][j][k] < crit.testSelectBinMins[k] || subnetResults[i][j][k] > crit.testSelectBinMaxes[k])
					testSelected = false;
				if (subnetResults[i][j][k] < crit.oppositeSelectBinMins[k] || subnetResults[i][j][k] > crit.oppositeSelectBinMaxes[k])
					oppositeSelected = false;
			}

			if (j < numSubnets) { //LONG
				if (testSelected)
					numLongTestSelected++;
				if (oppositeSelected)
					numShortOppositeSelected++;
			}
			else {
				if (testSelected)
					numShortTestSelected++;
				if (oppositeSelected)
					numLongOppositeSelected++;
			}
		}
		if (numLongTestSelected >= crit.minSubnetSelect && numLongOppositeSelected >= crit.oppositeMinSubnetSelect) {
			longProfit += (*dataset)[i].correctoutput;
			numLongTrades++;
			size_t binPos = 0;
			for (size_t j = 0; j < numBins; j++) {
				if ((*dataset)[i].correctbins[j] == BIN_POSITIVE_OUTPUT) {
					binPos = j;
					break;
				}
			}
			longDist[binPos]++;
		}
		if (numShortTestSelected >= crit.minSubnetSelect && numShortOppositeSelected >= crit.oppositeMinSubnetSelect) {
			shortProfit += (*dataset)[i].secondaryoutput;
			numShortTrades++;
			size_t binPos = 0;
			for (size_t j = 0; j < numBins; j++) {
				if ((*dataset)[i].secondarybins[j] == BIN_POSITIVE_OUTPUT) {
					binPos = j;
					break;
				}
			}
			shortDist[binPos]++;
		}
	}

	if (print) {
		std::cout << "Profits: L: " << longProfit << " (/" << numLongTrades << "=" << longProfit / numLongTrades << ") S: " << shortProfit << " (/" << numShortTrades << "=" << shortProfit / numShortTrades << ")" << " Total: " << longProfit + shortProfit << " (/" << numLongTrades + numShortTrades << "=" << (longProfit + shortProfit) / (numLongTrades + numShortTrades) << ")" << std::endl;
		std::cout << "Long Trade Distribution: ";
		for (size_t i = 0; i < numBins; i++) {
			std::cout << longDist[i] << " ";
		}
		std::cout << std::endl;

		std::cout << "Short Trade Distribution: ";
		for (size_t i = 0; i < numBins; i++) {
			std::cout << shortDist[i] << " ";
		}
		std::cout << std::endl;
	}

	float evaluation = 0.0f;
	float totalProfit = longProfit + shortProfit;
	size_t totalTrades = numLongTrades + numShortTrades;
	if (totalTrades != 0) {
		evaluation = (totalProfit) / fabs(totalProfit)*pow(totalProfit, selectionEvaluationProfitPower) / totalTrades;
	}

	if (print)
		std::cout << "Evaluation: " << evaluation << std::endl;

	return evaluation;
}

void loadLocalParameters(std::string parName) {
	std::ifstream infile(parName.c_str());
	std::string line;
	while (getline(infile, line)) {
		std::stringstream lss(line);
		std::string var;
		lss >> var;

		if (var == "numSubnets")
			lss >> numSubnets;
		else if (var == "minSubnetSelect")
			lss >> minSubnetSelect;
		else if (var == "oppositeMinSubnetSelect")
			lss >> oppositeMinSubnetSelect;
		else if (var == "longSelection")
			lss >> longSelection;
		else if (var == "shortSelection")
			lss >> shortSelection;
		else if (var == "testSelectBinMins") {
			while(!lss.eof()) {
				float binMin = -99999.0f;
				lss >> binMin;
				testSelectBinMins.push_back(binMin);
			}
		}
		else if (var == "testSelectBinMaxes") {
			while (!lss.eof()) {
				float binMax = 99999.0f;
				lss >> binMax;
				testSelectBinMaxes.push_back(binMax);
			}
		}
		else if (var == "oppositeSelectBinMins") {
			while(!lss.eof()) {
				float binMin = -99999.0f;
				lss >> binMin;
				oppositeSelectBinMins.push_back(binMin);
			}
		}
		else if (var == "oppositeSelectBinMaxes") {
			while (!lss.eof()) {
				float binMax = 99999.0f;
				lss >> binMax;
				oppositeSelectBinMaxes.push_back(binMax);
			}
		}
		else if (var == "randomizeSelectOptimizationStartPoint")
			lss >> randomizeSelectOptimizationStartPoint;
		else if (var == "testUseSampleFile")
			lss >> testUseSampleFile;
		else if (var == "testBegin")
			lss >> testBegin;
		else if (var == "testNumIOs")
			lss >> testNumIOs;
		else if (var == "discardSamples")
			lss >> discardSamples;
		else if (var == "testExplicitFile")
			lss >> testExplicitFile;
		else if (var == "selecttestfile")
			lss >> selecttestfile;
		else if (var == "selectionEvaluationType")
			lss >> selectionEvaluationType;
		else if (var == "selectionEvaluationProfitPower")
			lss >> selectionEvaluationProfitPower;
		else if (var == "selectPerturbSigma")
			lss >> selectPerturbSigma;
		else if (var == "selectBasedOnMaxBin")
			lss >> selectBasedOnMaxBin;
		else if (var == "binToSelectOn")
			lss >> binToSelectOn;
		else if (var == "numSelectCrossValSets")
			lss >> numSelectCrossValSets;
		else if (var == "selectOnMinBackupWeights")
			lss >> selectOnMinBackupWeights;
	}
}

//long outputs are stored in "correctoutput" and short in "secondaryoutput"
size_t readData(std::string fname, size_t begin, size_t numIOs) {
	if (numIOs > 0)
		std::cout << "Reading " << numIOs << " samples from data set " << fname << ": ";
	else
		std::cout << "Reading all samples from data set " << fname << ": ";

	auto readstart = std::chrono::high_resolution_clock::now();
	size_t totalSamples;
	totalSamples = readTwoPriceTrainSet(fname, begin, numIOs);

	auto readelapsed = std::chrono::high_resolution_clock::now() - readstart;
	long long readtime = std::chrono::duration_cast<std::chrono::microseconds>(readelapsed).count();
	std::cout << readtime / 1000000 << " s" << std::endl;
	size_t numSamples;
	if (numIOs > 0)
		numSamples = min(numIOs, totalSamples);
	else
		numSamples = totalSamples;
	std::cout << numSamples << "/" << totalSamples << " samples loaded" << std::endl;
	return numSamples;
}

SelectionCriteria getRandomSelectionCriteria() {
	SelectionCriteria crit;

	crit.minSubnetSelect = rand() % numSubnets + 1;
	crit.oppositeMinSubnetSelect = rand() % numSubnets + 1;
	crit.testSelectBinMins.resize(numBins);
	crit.testSelectBinMaxes.resize(numBins);
	crit.oppositeSelectBinMins.resize(numBins);
	crit.oppositeSelectBinMaxes.resize(numBins);
	for (size_t i = 0; i < numBins; i++) {
		int criteriaMin = -20;
		int criteriaMax = 125;
		if (rand() % 2 == 0) {
			crit.testSelectBinMins[i] = (float)(rand() % (criteriaMax - criteriaMin) + criteriaMin);
			crit.oppositeSelectBinMins[i] = (float)(rand() % (criteriaMax - criteriaMin) + criteriaMin);
			crit.testSelectBinMaxes[i] = rand() % (criteriaMax - (int)crit.testSelectBinMins[i]) + crit.testSelectBinMins[i];
			crit.oppositeSelectBinMaxes[i] = rand() % (criteriaMax - (int)crit.oppositeSelectBinMins[i]) + crit.oppositeSelectBinMins[i];
		}
		else {
			crit.testSelectBinMaxes[i] = (float)(rand() % (criteriaMax - criteriaMin) - criteriaMin);
			crit.oppositeSelectBinMaxes[i] = (float)(rand() % (criteriaMax - criteriaMin) - criteriaMin);
			crit.testSelectBinMins[i] = -(rand() % ((int)crit.testSelectBinMaxes[i] - criteriaMin)) + crit.testSelectBinMaxes[i];
			crit.oppositeSelectBinMins[i] = -(rand() % ((int)crit.oppositeSelectBinMaxes[i] - criteriaMin)) + crit.oppositeSelectBinMaxes[i];
		}
	}

	return crit;
}

SelectionCriteria perturbSelectionCriteria(SelectionCriteria crit) {
	size_t r = rand() % 4;
	if (r == 0 && crit.minSubnetSelect > 1)
		crit.minSubnetSelect--;
	else if (r == 1 && crit.minSubnetSelect < numSubnets)
		crit.minSubnetSelect++;

	r = rand() % 4;
	if (r == 0 && crit.oppositeMinSubnetSelect > 0)
		crit.oppositeMinSubnetSelect--;
	else if (r == 1 && crit.oppositeMinSubnetSelect < numSubnets)
		crit.oppositeMinSubnetSelect++;

	std::default_random_engine generator(rand());
	std::normal_distribution<double> distribution(0.0, selectPerturbSigma);

	for (size_t i = 0; i < numBins; i++) {
		crit.testSelectBinMins[i] += (float)distribution(generator);
		crit.testSelectBinMaxes[i] += (float)distribution(generator);
		crit.oppositeSelectBinMins[i] += (float)distribution(generator);
		crit.oppositeSelectBinMaxes[i] += (float)distribution(generator);
	}

	return crit;
}

void printSelectionCriteria(SelectionCriteria crit) {
	std::cout << "Min Subnets: T: " << crit.minSubnetSelect << " O: " << crit.oppositeMinSubnetSelect << std::endl;
	std::cout << "Test Bin Range: ";
	for (size_t i = 0; i < numBins; i++) {
		std::cout << "[" << crit.testSelectBinMins[i] << "," << crit.testSelectBinMaxes[i] << "] ";
	}
	std::cout << std::endl;
	std::cout << "Opposite Bin Range: ";
	for (size_t i = 0; i < numBins; i++) {
		std::cout << "[" << crit.oppositeSelectBinMins[i] << "," << crit.oppositeSelectBinMaxes[i] << "] ";
	}
	std::cout << std::endl;
}

void saveSelectionCriteria(SelectionCriteria crit) {
	std::stringstream css;
	css << savestring << savename << "sel";
	std::ofstream oss(css.str().c_str());

	oss << crit.minSubnetSelect << " " << crit.oppositeMinSubnetSelect << std::endl;
	for (size_t i = 0; i < numBins; i++) {
		oss << crit.testSelectBinMins[i] << " ";
	}
	oss << std::endl;
	for (size_t i = 0; i < numBins; i++) {
		oss << crit.testSelectBinMaxes[i] << " ";
	}
	oss << std::endl;
	for (size_t i = 0; i < numBins; i++) {
		oss << crit.oppositeSelectBinMins[i] << " ";
	}
	oss << std::endl;
	for (size_t i = 0; i < numBins; i++) {
		oss << crit.oppositeSelectBinMaxes[i] << " ";
	}
	oss << std::endl;
}

bool loadSelectionCriteria(SelectionCriteria* crit) {
	std::stringstream css;
	css << savestring << savename << "sel";

	if (!PathFileExists(css.str().c_str())) {
		std::cout << "No criteria file found" << std::endl;
		return false;
	}

	std::ifstream iss(css.str().c_str());

	iss >> crit->minSubnetSelect >> crit->oppositeMinSubnetSelect;
	crit->testSelectBinMins.resize(numBins);
	crit->testSelectBinMaxes.resize(numBins);
	crit->oppositeSelectBinMins.resize(numBins);
	crit->oppositeSelectBinMaxes.resize(numBins);
	for (size_t i = 0; i < numBins; i++) {
		iss >> crit->testSelectBinMins[i];
	}
	for (size_t i = 0; i < numBins; i++) {
		iss >> crit->testSelectBinMaxes[i];
	}
	for (size_t i = 0; i < numBins; i++) {
		iss >> crit->oppositeSelectBinMins[i];
	}
	for (size_t i = 0; i < numBins; i++) {
		iss >> crit->oppositeSelectBinMaxes[i];
	}
	return true;
}

//don't do opposite selecting right now
void evaluateMaxBinSelection(size_t bin, size_t numSubnetsToSelect, bool print) {
	std::vector<IOPair>* dataset = getTrainSet();
	float longProfit = 0.0f;
	float shortProfit = 0.0f;
	size_t numLongTrades = 0;
	size_t numShortTrades = 0;
	std::vector<size_t> longDist(numBins);
	std::vector<size_t> shortDist(numBins);
	for (size_t i = 0; i < subnetResults.size(); i++) {
		size_t numLongTestSelected = 0;
		size_t numShortTestSelected = 0;
		for (size_t j = 0; j < 2 * numSubnets; j++) {
			float maxBinWeight = 0.0f;
			size_t maxBin = 0;
			for (size_t k = 0; k < numBins; k++) {
				if (subnetResults[i][j][k] > maxBinWeight) {
					maxBinWeight = subnetResults[i][j][k];
					maxBin = k;
				}
			}

			if (j < numSubnets) { //LONG
				if (maxBinWeight > 0.0f && maxBin == bin)
					numLongTestSelected++;
			}
			else {
				if (maxBinWeight > 0.0f && maxBin == bin)
					numShortTestSelected++;
			}
		}
		if (numLongTestSelected >= numSubnetsToSelect) {
			longProfit += (*dataset)[i].correctoutput;
			numLongTrades++;
			size_t binPos = 0;
			for (size_t j = 0; j < numBins; j++) {
				if ((*dataset)[i].correctbins[j] == BIN_POSITIVE_OUTPUT) {
					binPos = j;
					break;
				}
			}
			longDist[binPos]++;
		}
		if (numShortTestSelected >= numSubnetsToSelect) {
			shortProfit += (*dataset)[i].secondaryoutput;
			numShortTrades++;
			size_t binPos = 0;
			for (size_t j = 0; j < numBins; j++) {
				if ((*dataset)[i].secondarybins[j] == BIN_POSITIVE_OUTPUT) {
					binPos = j;
					break;
				}
			}
			shortDist[binPos]++;
		}
	}

	if (print) {
		std::cout << "Bin #" << bin << " with " << numSubnetsToSelect << " nets required. ";
		std::cout << "Profits: L: " << longProfit << " (/" << numLongTrades << "=" << longProfit / numLongTrades << ") S: " << shortProfit << " (/" << numShortTrades << "=" << shortProfit / numShortTrades << ")" << " Total: " << longProfit + shortProfit << " (/" << numLongTrades + numShortTrades << "=" << (longProfit + shortProfit) / (numLongTrades + numShortTrades) << ")" << std::endl;
		std::cout << "Long Trade Distribution: ";
		for (size_t i = 0; i < numBins; i++) {
			std::cout << longDist[i] << " ";
		}
		std::cout << std::endl;

		std::cout << "Short Trade Distribution: ";
		for (size_t i = 0; i < numBins; i++) {
			std::cout << shortDist[i] << " ";
		}
		std::cout << std::endl;
	}
}